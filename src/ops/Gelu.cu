#include "hip/hip_runtime.h"
#include "gpu_runtime.h"
#define pi 3.14159265358979323846
#define e  2.71828182845904523536

__global__ void Gelu_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = input[ind] * 0.5 * (1.0 + erf( input[ind]/sqrt(2.0)));
}

int DLGpuGelu(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        Gelu_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        Gelu_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}

__global__ void gelu_grad_kernel(const float *input, const float *in_grad,
                                 float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = in_grad[ind]*(0.5+0.5*erf( input[ind]/sqrt(2.0))+0.5*input[ind]*(sqrt(2.0)*pow(e,(-0.5*pow(input[ind],2)))/sqrt(pi)));
}

int DLGpuGeluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output,
                      DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    const float *input_data = (const float *)input->data;
    const float *in_grad_data = (const float *)in_grad->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        gelu_grad_kernel<<<blocks, threads, 0,
                           *(hipStream_t *)stream_handle->handle>>>(
            input_data, in_grad_data, output_data, size);
    else
        gelu_grad_kernel<<<blocks, threads>>>(input_data, in_grad_data,
                                              output_data, size);
    return 0;
}
