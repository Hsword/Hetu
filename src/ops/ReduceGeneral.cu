#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

int DLGpuReduceGeneral(hipdnnReduceTensorOp_t red_type,
                       const DLArrayHandle in_arr, DLArrayHandle out_arr,
                       int *axes, int num_ax,
                       DLStreamHandle stream_handle = NULL, int offset = 0) {
    int dev_id = (in_arr->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    float one = 1.0f;
    float zero = 0.0f;
    hipdnnReduceTensorDescriptor_t rtd;
    CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
    CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, red_type, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES));

    hipdnnTensorDescriptor_t adesc;
    hipdnnTensorDescriptor_t cdesc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

    int ori_ndim = in_arr->ndim;
    int ndim = max(ori_ndim, 4);
    size_t cpu_mem = ndim * sizeof(int);
    int *dimA = (int *)malloc(cpu_mem);
    int *strideA = (int *)malloc(cpu_mem);
    int *dimC = (int *)malloc(cpu_mem);
    int *strideC = (int *)malloc(cpu_mem);

    for (int i = 0; i < ori_ndim; ++i) {
        dimA[i] = dimC[i] = (int)in_arr->shape[i];
    }
    for (int i = ori_ndim; i < ndim; ++i) {
        dimA[i] = dimC[i] = 1;
    }
    for (int i = 0; i < num_ax; ++i) {
        assert(axes[i] < ori_ndim && axes[i] >= 0);
        dimC[axes[i]] = 1;
    }
    int temp_strideA = 1;
    int temp_strideC = 1;
    for (int i = ndim - 1; i >= 0; --i) {
        strideA[i] = temp_strideA;
        strideC[i] = temp_strideC;
        temp_strideA *= dimA[i];
        temp_strideC *= dimC[i];
    }

    size_t size = temp_strideA * sizeof(float);

    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *workspace = find_chunk(size, dev_id);

    CUDNN_CALL(hipdnnSetTensorNdDescriptor(adesc, HIPDNN_DATA_FLOAT, ndim, dimA,
                                          strideA));
    CUDNN_CALL(hipdnnSetTensorNdDescriptor(cdesc, HIPDNN_DATA_FLOAT, ndim, dimC,
                                          strideC));
    void *out_ptr = (void *)out_arr->data;
    if (offset > 0) {
        size_t fsize = offset * sizeof(float);
        out_ptr = out_ptr + fsize;
    }
    CUDNN_CALL(hipdnnReduceTensor(cudnn_map[dev_id], rtd, NULL, 0, workspace,
                                 size, &one, adesc, (const void *)in_arr->data,
                                 &zero, cdesc, out_ptr));

    del_chunk(workspace, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(adesc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(cdesc));
    CUDNN_CALL(hipdnnDestroyReduceTensorDescriptor(rtd));
    free(dimA);
    free(dimC);
    free(strideA);
    free(strideC);
    return 0;
}

int DLGpuReduceMean(const DLArrayHandle in_arr, DLArrayHandle out_arr,
                    int *axes, int num_ax,
                    DLStreamHandle stream_handle = NULL) {
    return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_AVG, in_arr, out_arr, axes,
                              num_ax, stream_handle);
}

// int DLGpuReduceSum(const DLArrayHandle in_arr, DLArrayHandle out_arr, int *axes,
//                    int num_ax, DLStreamHandle stream_handle = NULL) {
//     return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_ADD, in_arr, out_arr, axes,
//                               num_ax, stream_handle);
// }

int DLGpuReduceMul(const DLArrayHandle in_arr, DLArrayHandle out_arr, int *axes,
                   int num_ax, DLStreamHandle stream_handle = NULL) {
    return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_MUL, in_arr, out_arr, axes,
                              num_ax, stream_handle);
}

int DLGpuReduceMin(const DLArrayHandle in_arr, DLArrayHandle out_arr, int *axes,
                   int num_ax, DLStreamHandle stream_handle = NULL) {
    return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_MIN, in_arr, out_arr, axes,
                              num_ax, stream_handle);
}

int DLGpuReduceNorm1(const DLArrayHandle in_arr, DLArrayHandle out_arr,
                     int *axes, int num_ax,
                     DLStreamHandle stream_handle = NULL) {
    return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_NORM1, in_arr, out_arr, axes,
                              num_ax, stream_handle);
}

int DLGpuReduceNorm2(const DLArrayHandle in_arr, DLArrayHandle out_arr,
                     int *axes, int num_ax,
                     DLStreamHandle stream_handle = NULL) {
    return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_NORM2, in_arr, out_arr, axes,
                              num_ax, stream_handle);
}

int DLGpuReduceNorm2Raw(const DLArrayHandle in_arr, DLArrayHandle out_arr,
                        int *axes, int num_ax, int offset = 0,
                        DLStreamHandle stream_handle = NULL) {
    if (in_arr->ndim == 1 && in_arr->shape[0] == 1) {
        size_t fsize = sizeof(float);
        void *out_ptr = (void *)out_arr->data;
        if (offset > 0) {
            size_t foffset = fsize * offset;
            out_ptr = out_ptr + foffset;
        }
        hipStream_t cu_stream = static_cast<hipStream_t>(
            stream_handle ? *(hipStream_t *)(stream_handle->handle) : NULL);
        hipMemcpyAsync((void *)in_arr->data, out_ptr, fsize,
                        hipMemcpyDeviceToDevice, cu_stream);
        return 0;
    } else {
        return DLGpuReduceGeneral(HIPDNN_REDUCE_TENSOR_NORM2, in_arr, out_arr,
                                  axes, num_ax, stream_handle, offset);
    }
}
