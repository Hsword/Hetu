#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void sigmoid_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = 1.0f / (1.0f + exp(-input[ind]));
}

int DLGpuSigmoid(const DLArrayHandle input, DLArrayHandle output,
                 DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        sigmoid_kernel<<<blocks, threads, 0,
                         *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        sigmoid_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}