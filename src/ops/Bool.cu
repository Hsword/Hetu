#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void bool_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    if(input[ind] > 0){
        output[ind] = 1;
    }
    else{
        output[ind] = 0;
    }
}

int DLGpuBool(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        bool_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        bool_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}