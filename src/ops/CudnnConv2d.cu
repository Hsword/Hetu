#include "gpu_runtime.h"

int CuDNN_DLGpuConv2d(const DLArrayHandle input_x, const DLArrayHandle input_f,
                      DLArrayHandle output, const int padding_h,
                      const int padding_w, const int stride_h,
                      const int stride_w, DLStreamHandle stream_handle = NULL) {
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    // input
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    // filter
    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));

    // convolution
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    size_t out_N = output->shape[0];
    size_t out_C = output->shape[1];
    size_t out_H = output->shape[2];
    size_t out_W = output->shape[3];
    // output
    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, out_N, out_C, out_H,
                                          out_W));
    float *output_data = (float *)output->data;

    // search for the best algorithm
    int request_cnt = 9, return_cnt = 9;
    hipdnnConvolutionFwdAlgoPerf_t algo_perf[9];
    CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc,
        request_cnt, &return_cnt, algo_perf));

    if (is_chunk_init(dev_id) == false)
        chunk_init(dev_id);

    size_t workspace_size;
    void *work_data = nullptr;
    hipdnnConvolutionFwdAlgo_t algo;
    for(int i = 0; i < return_cnt; ++i) {
        CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
            cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc, algo_perf[i].algo,
            &workspace_size));
        work_data = find_chunk(workspace_size, dev_id, false);
        if (work_data) {
            algo = algo_perf[i].algo;
            break;
        }
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn_map[dev_id], &alpha, input_desc, input_data, filter_desc,
        filter_data, conv_desc, algo, work_data, workspace_size, &beta,
        out_desc, output_data));
    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    return 0;
}
int CuDNN_DLGpuConv2d_Gradient_of_Filter(const DLArrayHandle input_x,
                                         const DLArrayHandle gradient_y,
                                         DLArrayHandle gradient_f,
                                         const int padding_h,
                                         const int padding_w,
                                         const int stride_h, const int stride_w,
                                         DLStreamHandle stream_handle = NULL) {
    // create handle
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    // input
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    // dy
    size_t dy_N = gradient_y->shape[0];
    size_t dy_C = gradient_y->shape[1];
    size_t dy_H = gradient_y->shape[2];
    size_t dy_W = gradient_y->shape[3];
    const float *dy_data = (const float *)gradient_y->data;

    hipdnnTensorDescriptor_t dy_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dy_N, dy_C, dy_H, dy_W));

    // conv2d
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    // dw
    size_t df_N = gradient_f->shape[0];
    size_t df_C = gradient_f->shape[1];
    size_t df_H = gradient_f->shape[2];
    size_t df_W = gradient_f->shape[3];
    float *df_data = (float *)gradient_f->data;

    hipdnnFilterDescriptor_t df_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&df_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        df_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, df_N, df_C, df_H, df_W));

    // search for the best algorithm
    int request_cnt = 9, return_cnt = 9;
    hipdnnConvolutionBwdFilterAlgoPerf_t algo_perf[9];
    CUDNN_CALL(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
        cudnn_map[dev_id], input_desc, dy_desc, conv_desc, df_desc,
        request_cnt, &return_cnt, algo_perf));

    hipError_t err;
    size_t workspace_size;
    void *work_data = nullptr;
    hipdnnConvolutionBwdFilterAlgo_t algo;
    for(int i = 0; i < return_cnt; ++i) {
        CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
            cudnn_map[dev_id], input_desc, dy_desc, conv_desc, df_desc, algo_perf[i].algo,
            &workspace_size));
        err = hipMalloc(&work_data, workspace_size);
        if (err == hipSuccess) {
            algo = algo_perf[i].algo;
            break;
        }
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(
        cudnn_map[dev_id], &alpha, input_desc, input_data, dy_desc, dy_data,
        conv_desc, algo, work_data, workspace_size, &beta, df_desc, df_data));
    hipFree(work_data);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(df_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    return 0;
}

int CuDNN_DLGpuConv2d_Gradient_of_Data(const DLArrayHandle input_f,
                                       const DLArrayHandle gradient_y,
                                       DLArrayHandle gradient_x,
                                       const int padding_h, const int padding_w,
                                       const int stride_h, const int stride_w,
                                       DLStreamHandle stream_handle = NULL) {
    // create handle
    int dev_id = (input_f->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    // filter
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));
    // dy
    size_t dy_N = gradient_y->shape[0];
    size_t dy_C = gradient_y->shape[1];
    size_t dy_H = gradient_y->shape[2];
    size_t dy_W = gradient_y->shape[3];
    const float *dy_data = (const float *)gradient_y->data;

    hipdnnTensorDescriptor_t dy_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dy_N, dy_C, dy_H, dy_W));

    // conv2d
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    // dx
    size_t dx_N = gradient_x->shape[0];
    size_t dx_C = gradient_x->shape[1];
    size_t dx_H = gradient_x->shape[2];
    size_t dx_W = gradient_x->shape[3];
    float *dx_data = (float *)gradient_x->data;

    hipdnnTensorDescriptor_t dx_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dx_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dx_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dx_N, dx_C, dx_H, dx_W));

    // search for the best algorithm
    int request_cnt = 9, return_cnt = 9;
    hipdnnConvolutionBwdDataAlgoPerf_t algo_perf[9];
    CUDNN_CALL(cudnnGetConvolutionBackwardDataAlgorithm_v7(
        cudnn_map[dev_id], filter_desc, dy_desc, conv_desc, dx_desc,
        request_cnt, &return_cnt, algo_perf));

    hipError_t err;
    size_t workspace_size;
    void *work_data = nullptr;
    hipdnnConvolutionBwdDataAlgo_t algo;
    for(int i = 0; i < return_cnt; ++i) {
        CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
            cudnn_map[dev_id], filter_desc, dy_desc, conv_desc, dx_desc, algo_perf[i].algo,
            &workspace_size));
        err = hipMalloc(&work_data, workspace_size);
        if (err == hipSuccess) {
            algo = algo_perf[i].algo;
            break;
        }
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardData(
        cudnn_map[dev_id], &alpha, filter_desc, filter_data, dy_desc, dy_data,
        conv_desc, algo, work_data, workspace_size, &beta, dx_desc, dx_data));
    hipFree(work_data);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dx_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    return 0;
}
