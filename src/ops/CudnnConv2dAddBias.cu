#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void conv2d_add_bias(size_t nthreads,
    const float *input_data,
    float *output_data,
    size_t input_size,
    size_t output_size) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nthreads)
    return;
    size_t input_id = id % input_size / output_size;
    output_data[id] += input_data[input_id];
}


int Cudnn_Conv2dAddBias(const DLArrayHandle input_x, const DLArrayHandle input_f,
                      const DLArrayHandle bias, DLArrayHandle output,
                      const int padding_h, const int padding_w,
                      const int stride_h, const int stride_w,
                      DLStreamHandle stream_handle = NULL) {
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    // input
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    // filter
    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));

    // convolution
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    size_t out_N = output->shape[0];
    size_t out_C = output->shape[1];
    size_t out_H = output->shape[2];
    size_t out_W = output->shape[3];
    // output
    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, out_N, out_C, out_H,
                                          out_W));
    float *output_data = (float *)output->data;

    // search for the best algorithm
    int request_cnt = 9, return_cnt = 9;
    hipdnnConvolutionFwdAlgoPerf_t algo_perf[9];
    CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc,
        request_cnt, &return_cnt, algo_perf));

    if (is_chunk_init(dev_id) == false)
        chunk_init(dev_id);

    size_t workspace_size;
    void *work_data = nullptr;
    hipdnnConvolutionFwdAlgo_t algo;
    for(int i = 0; i < return_cnt; ++i) {
        CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
            cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc, algo_perf[i].algo,
            &workspace_size));
        work_data = find_chunk(workspace_size, dev_id, false);
        if (work_data) {
            algo = algo_perf[i].algo;
            break;
        }
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn_map[dev_id], &alpha, input_desc, input_data, filter_desc,
        filter_data, conv_desc, algo, work_data, workspace_size, &beta,
        out_desc, output_data));

    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    
    // add bias
    const float *bias_data = (const float*)bias->data;
    size_t nthreads = out_N * out_C * out_H * out_W;
    size_t BLOCKS = (nthreads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    size_t bias_output_size = out_H * out_W;
    size_t bias_input_size = out_C * bias_output_size;
    if (stream_handle)
        conv2d_add_bias<<<BLOCKS, THREADS_PER_BLOCK, 0,
                                     *(hipStream_t *)stream_handle->handle>>>(
            nthreads, bias_data, output_data, bias_input_size, bias_output_size);
    else
        conv2d_add_bias<<<BLOCKS, THREADS_PER_BLOCK>>>(
            nthreads, bias_data, output_data, bias_input_size, bias_output_size);
    return 0;
}
