#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

int DLGpuAddmm(const DLArrayHandle input, const DLArrayHandle matA,
               const DLArrayHandle matB, float alpha, float beta,
               DLArrayHandle matC, DLStreamHandle stream_handle = NULL) {
    // cublas assume matrix is column major
    assert(input->ndim == 2);
    assert(matA->ndim == 2);
    assert(matB->ndim == 2);
    assert(matC->ndim == 2);

    int dev_id = (input->ctx).device_id;
    cublas_init(dev_id, stream_handle);

    int m = matC->shape[1];
    int n = matC->shape[0];
    int k = matA->shape[1];

    float *input_data = (float *)input->data;
    float *output_data = (float *)matC->data;
    int size = 1;
    for (int i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }

    hipMemcpy((void *)output_data, (void *)input_data, size * sizeof(float),
               hipMemcpyDeviceToDevice);

    hipblasSgemm(cublas_map[dev_id], HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                (const float *)matB->data, m, (const float *)matA->data, k,
                &beta, (float *)output_data, m);
    return 0;
}

int DLGpuAddmmGradient(const DLArrayHandle input, DLArrayHandle output,
                       int axis, float beta,
                       DLStreamHandle stream_handle = NULL) {
    int dev_id = (input->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    float zero = 0.0f;
    hipdnnReduceTensorDescriptor_t rtd;
    CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
    CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES));

    hipdnnTensorDescriptor_t adesc;
    hipdnnTensorDescriptor_t cdesc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

    int ori_ndim = input->ndim;
    int ndim = max(ori_ndim, 4);
    size_t cpu_mem = ndim * sizeof(int);
    int *dimA = (int *)malloc(cpu_mem);
    int *strideA = (int *)malloc(cpu_mem);
    int *dimC = (int *)malloc(cpu_mem);
    int *strideC = (int *)malloc(cpu_mem);

    for (int i = 0; i < ori_ndim; ++i) {
        dimA[i] = dimC[i] = (int)input->shape[i];
    }
    for (int i = ori_ndim; i < ndim; ++i) {
        dimA[i] = dimC[i] = 1;
    }
    dimC[0] = 1;
    int temp_strideA = 1;
    int temp_strideC = 1;
    for (int i = ndim - 1; i >= 0; --i) {
        strideA[i] = temp_strideA;
        strideC[i] = temp_strideC;
        temp_strideA *= dimA[i];
        temp_strideC *= dimC[i];
    }

    size_t size = temp_strideA * sizeof(float);

    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *workspace = find_chunk(size, dev_id);

    CUDNN_CALL(hipdnnSetTensorNdDescriptor(adesc, HIPDNN_DATA_FLOAT, ndim, dimA,
                                          strideA));
    CUDNN_CALL(hipdnnSetTensorNdDescriptor(cdesc, HIPDNN_DATA_FLOAT, ndim, dimC,
                                          strideC));
    CUDNN_CALL(hipdnnReduceTensor(cudnn_map[dev_id], rtd, NULL, 0, workspace,
                                 size, &beta, adesc, (const void *)input->data,
                                 &zero, cdesc, (void *)output->data));

    del_chunk(workspace, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(adesc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(cdesc));
    CUDNN_CALL(hipdnnDestroyReduceTensorDescriptor(rtd));
    free(dimA);
    free(dimC);
    free(strideA);
    free(strideC);
    return 0;
}
