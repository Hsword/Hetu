#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void pow_kernel(const float *input, float *output, float exp,
                           size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = pow(input[ind], exp);
}

int DLGpuPow(const DLArrayHandle input, DLArrayHandle output, float exp,
             DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        pow_kernel<<<blocks, threads, 0,
                     *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, exp, size);
    else
        pow_kernel<<<blocks, threads>>>(input_data, output_data, exp, size);
    return 0;
}

__global__ void pow_grad_kernel(const float *input, const float *in_grad,
                                float *output, float exp, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = exp * pow(input[ind], (exp - 1)) * in_grad[ind];
}

int DLGpuPowGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                     DLArrayHandle output, float exp,
                     DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    const float *input_data = (const float *)input->data;
    const float *in_grad_data = (const float *)in_grad->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        pow_grad_kernel<<<blocks, threads, 0,
                          *(hipStream_t *)stream_handle->handle>>>(
            input_data, in_grad_data, output_data, exp, size);
    else
        pow_grad_kernel<<<blocks, threads>>>(input_data, in_grad_data,
                                             output_data, exp, size);
    return 0;
}
