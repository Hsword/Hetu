#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void tanh_kernel(const float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = tanhf(input[ind]);
}

int DLGpuTanh(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    const float *input_data = (const float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        tanh_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        tanh_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}

__global__ void tanh_gradient_kernel(const float *forward, const float *grad,
                                     float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (1 - forward[ind] * forward[ind]) * grad[ind];
}

int DLGpuTanhGradient(const DLArrayHandle forward, const DLArrayHandle grad,
                      DLArrayHandle output,
                      DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < forward->ndim; i++) {
        size *= forward->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    const float *forward_data = (const float *)forward->data;
    const float *grad_data = (const float *)grad->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        tanh_gradient_kernel<<<blocks, threads, 0,
                               *(hipStream_t *)stream_handle->handle>>>(
            forward_data, grad_data, output_data, size);
    else
        tanh_gradient_kernel<<<blocks, threads>>>(forward_data, grad_data,
                                                  output_data, size);
    return 0;
}