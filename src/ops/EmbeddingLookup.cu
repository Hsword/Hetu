#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void embedding_lookup_kernel(const float *input, const float *ids,
                                        float *output, size_t size,
                                        size_t length, size_t input_row) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;
    int id = ids[index];
    float *output_ptr = output + length * index;
    if (id < 0 || id >= input_row) {
        for (int i = 0; i < length; i++)
            output_ptr[i] = 0;
    } else {
        const float *input_ptr = input + length * id;
        for (int i = 0; i < length; i++)
            output_ptr[i] = input_ptr[i];
    }
}

int DLGpuEmbeddingLookUp(const DLArrayHandle input, const DLArrayHandle ids,
                         DLArrayHandle output,
                         DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = 1;
    for (int i = 0; i < output->ndim; i++) {
        if (i < output->ndim - 1) {
            assert(ids->shape[i] == output->shape[i]);
        } else if (i == output->ndim - 1) {
            assert(input->shape[1] == output->shape[i]);
        }
    }
    for (int i = 0; i < ids->ndim; i++) {
        size = size * ids->shape[i];
    }
    size_t input_row = input->shape[0];
    size_t length = input->shape[1];
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    const float *id_list = (const float *)ids->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        embedding_lookup_kernel<<<blocks, threads, 0,
                                  *(hipStream_t *)stream_handle->handle>>>(
            input_data, id_list, output_data, size, length, input_row);
    else
        embedding_lookup_kernel<<<blocks, threads>>>(input_data, id_list,
                                                     output_data, size, length, input_row);
    return 0;
}

__global__ void array_set_zero_kernel(float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = 0;
}

__global__ void embedding_lookup_gradient_kernel(const float *output_grad_data,
                                                 const float *ids,
                                                 float *input_grad_data,
                                                 size_t size, size_t length) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;
    int id = ids[index];
    const float *output_grad_ptr = output_grad_data + length * index;
    float *input_grad_ptr = input_grad_data + length * id;
    for (int i = 0; i < length; i++)
        atomicAdd(input_grad_ptr + i, *(output_grad_ptr + i));
}

int DLGpuEmbeddingLookUp_Gradient(const DLArrayHandle output_grad,
                                  const DLArrayHandle ids,
                                  DLArrayHandle input_grad,
                                  DLStreamHandle stream_handle = NULL) {
    assert(input_grad->ndim == 2);
    size_t size = 1;
    for (int i = 0; i < output_grad->ndim; i++) {
        if (i < output_grad->ndim - 1) {
            assert(ids->shape[i] == output_grad->shape[i]);
        } else if (i == output_grad->ndim - 1) {
            assert(input_grad->shape[1] == output_grad->shape[i]);
        }
    }
    for (int i = 0; i < ids->ndim; i++) {
        size = size * ids->shape[i];
    }
    size_t length = input_grad->shape[1];
    dim3 blocks;
    dim3 threads;
    const float *output_grad_data = (const float *)output_grad->data;
    float *input_grad_data = (float *)input_grad->data;
    const float *id_list = (const float *)ids->data;

    size_t input_grad_size = 1;
    for (int i = 0; i < input_grad->ndim; i++) {
        input_grad_size *= input_grad->shape[i];
    }
    if (input_grad_size <= 1024) {
        threads.x = input_grad_size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (input_grad_size + 1023) / 1024;
    }
    if (stream_handle)
        array_set_zero_kernel<<<blocks, threads, 0,
                                *(hipStream_t *)stream_handle->handle>>>(
            input_grad_data, input_grad_size);
    else
        array_set_zero_kernel<<<blocks, threads>>>(input_grad_data,
                                                   input_grad_size);

    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        embedding_lookup_gradient_kernel<<<
            blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
            output_grad_data, id_list, input_grad_data, size, length);
    else
        embedding_lookup_gradient_kernel<<<blocks, threads>>>(
            output_grad_data, id_list, input_grad_data, size, length);
    return 0;
}