#include "../header/mpi_nccl_communication.h"

static const ncclDataType_t TYPE2TYPE_V1[] = {
    ncclChar,    // ncclInt8, ncclChar
    ncclUint8,   // ncclUint8
    ncclInt32,   // ncclInt32, ncclInt
    ncclUint32,  // ncclUint32
    ncclInt64,   // ncclInt64
    ncclUint64,  // ncclUint64
    ncclFloat16, // ncclFloat16, ncclHalf
    ncclFloat32, // ncclFloat32, ncclFloat
    ncclFloat64  // ncclFloat64, ncclDouble
};

ncclDataType_t _get_proper_datatype(int datatype) {
    return TYPE2TYPE_V1[datatype];
}

#if NCCL_MINOR >= 10
static const ncclRedOp_t TYPE2TYPE_V2[] = {ncclSum, ncclProd, ncclMax, ncclMin,
                                           ncclAvg};
#else
static const ncclRedOp_t TYPE2TYPE_V2[] = {ncclSum, ncclProd, ncclMax, ncclMin};
#endif

ncclRedOp_t _get_proper_redop(int redop) {
    assert(redop < sizeof(TYPE2TYPE_V2) / sizeof(ncclRedOp_t));
    return TYPE2TYPE_V2[redop];
}

void MPIInit() {
    MPICHECK(MPI_Init(NULL, NULL));
}

void MPIFinalize() {
    MPICHECK(MPI_Finalize());
}

void MPIGetComm(MPI_Comm *comm) {
    *comm = MPI_COMM_WORLD;
}

void MPIBcast(void *buffer, int size, int root, MPI_Comm comm) {
    MPICHECK(MPI_Bcast(buffer, size, MPI_BYTE, root, comm));
}

void getMPICommRank(MPI_Comm *comm, int *myRank) {
    MPICHECK(MPI_Comm_rank(*comm, myRank));
}

void getMPICommSize(MPI_Comm *comm, int *nRanks) {
    MPICHECK(MPI_Comm_size(*comm, nRanks));
}

uint64_t getHostHash(const char *string) {
    // Based on DJB2, result = result * 33 + char
    uint64_t result = 5381;
    for (int c = 0; string[c] != '\0'; c++) {
        result = (((result << 5) + result) + string[c]) % 1000003;
    }
    return result;
}

void getHostName(char *hostname, int maxlen) {
    gethostname(hostname, maxlen);
    for (int i = 0; i < maxlen; i++) {
        if (hostname[i] == '.') {
            hostname[i] = '\0';
            return;
        }
    }
}

void getLocalRank(MPI_Comm *comm, int nRanks, int myRank, int *localRank,
                  unsigned long long hostHashs[]) {
    int _localRank = 0;
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                           sizeof(unsigned long long), MPI_BYTE, *comm));
    for (int p = 0; p < nRanks; p++) {
        if (p == myRank)
            break;
        if (hostHashs[p] == hostHashs[myRank])
            (_localRank)++;
    }
    *localRank = _localRank;
}

void getGlobalDevice(MPI_Comm *comm, int nRanks, int myRank, int device_id,
                     int hostDevices[]) {
    hostDevices[myRank] = device_id;
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostDevices,
                           sizeof(int), MPI_BYTE, *comm));
}

void setDevice(int device_id) {
    CUDACHECK(hipSetDevice(device_id));
}

void getNcclUniqueId(ncclUniqueId *Id, MPI_Comm mpi_comm, int localRank,
                     int senderRank) {
    if (localRank == 0)
        NCCLCHECK(ncclGetUniqueId(Id));
    MPIBcast((void *)Id, sizeof(ncclUniqueId), senderRank, mpi_comm);
}

void getGroupNcclUniqueId(ncclUniqueId *Id, MPI_Comm mpi_comm, int rank,
                          int dests[], int group_size, int group_id) {
    // we assume that group size >= 2
    if (dests[0] == rank) {
        NCCLCHECK(ncclGetUniqueId(Id));
        for (int i = 1; i < group_size; ++i) {
            MPICHECK(MPI_Send((const void *)Id, sizeof(ncclUniqueId), MPI_BYTE,
                              dests[i], group_id, mpi_comm));
        }
    } else {
        MPICHECK(MPI_Recv((void *)Id, sizeof(ncclUniqueId), MPI_BYTE, dests[0],
                          group_id, mpi_comm, MPI_STATUS_IGNORE));
    }
}

void initNcclCommRank(ncclComm_t *comm, int nranks, ncclUniqueId *commId,
                      int rank, int localRank) {
    NCCLCHECK(ncclCommInitRank(comm, nranks, *commId, rank));
}

void GroupStart() {
    NCCLCHECK(ncclGroupStart());
}

void GroupEnd() {
    NCCLCHECK(ncclGroupEnd());
}

void _ncclAllReduce(const void *sendbuff, void *recvbuff, int size,
                    int datatype, int op, ncclComm_t comm,
                    hipStream_t stream) {
    NCCLCHECK(ncclAllReduce((const void *)sendbuff, (void *)recvbuff, size,
                            _get_proper_datatype(datatype),
                            _get_proper_redop(op), comm, stream));
}

void _ncclReduce(const void *sendbuff, void *recvbuff, size_t count,
                 int datatype, int op, int root, ncclComm_t comm,
                 hipStream_t stream) {
    NCCLCHECK(ncclReduce(sendbuff, recvbuff, count,
                         _get_proper_datatype(datatype), _get_proper_redop(op),
                         root, comm, stream));
}
void _ncclHAllToAll(const void *sendbuff, void *recvbuff, int size, int datatype, ncclComm_t comm, hipStream_t stream, int num_nodes, int num_local_gpus) {
    ncclDataType_t type = _get_proper_datatype(datatype);
    int unit_size = sizeof(type);
    switch(type){
        case 0: 
        case 1:   
            unit_size = 1;   
            break;                  
        case 5:      
            unit_size = 2;   
            break;                  
        case 2:          
        case 3:       
        case 7:   
            unit_size = 4;   
            break;                  
        case 4:    
        case 8:   
            unit_size = 8;
            break;                      
    }
    int split_size = size / num_nodes;
    GroupStart();
    for(int i = 0; i < num_nodes; i++){
        NCCLCHECK(ncclSend(sendbuff+i*split_size*unit_size, split_size, type, i*num_local_gpus, comm, stream));
        NCCLCHECK(ncclRecv(recvbuff+i*split_size*unit_size, split_size, type, i*num_local_gpus, comm, stream));
    }
    GroupEnd();
}

void _ncclHA2AGather(const void *send_buff, void *recv_buff, int size, int datatype, int myrank, int num_local_gpus, ncclComm_t comm, hipStream_t stream){
    ncclDataType_t type = _get_proper_datatype(datatype);
    int unit_size = sizeof(type);
    switch(type){               
        case 0:         
        case 1:   
            unit_size = 1;   
            break;                  
        case 5:   
            unit_size = 2;   
            break;                  
        case 2:          
        case 3:       
        case 7:   
            unit_size = 4;   
            break;                  
        case 4:          
        case 8:  
            unit_size = 8;   
            break;        
    } 
    GroupStart(); 
    if(myrank%num_local_gpus==0){
        for(int i=0; i<num_local_gpus; i++){
            NCCLCHECK(ncclRecv(recv_buff+i*size*unit_size, size, type, i+myrank, comm, stream));
        }
    }
    int target = myrank - myrank%num_local_gpus;
    NCCLCHECK(ncclSend(send_buff, size, type, target, comm, stream));
    GroupEnd();
}

void _ncclHA2AScatter(const void *send_buff, void *recv_buff, int size, int datatype, int myrank, int num_local_gpus, ncclComm_t comm, hipStream_t stream){
   ncclDataType_t type = _get_proper_datatype(datatype);
    int unit_size = sizeof(type);
    switch(type){               
        case 0:         
        case 1:   
            unit_size = 1;   
            break;                  
        case 5:   
            unit_size = 2;   
            break;                  
        case 2:          
        case 3:       
        case 7:   
            unit_size = 4;   
            break;                  
        case 4:          
        case 8:  
            unit_size = 8;   
            break;        
    }   
    GroupStart(); 
    if(myrank%num_local_gpus==0){
        for(int i=0; i<num_local_gpus; i++){
            NCCLCHECK(ncclSend(send_buff+i*size*unit_size, size, type, i+myrank, comm, stream));
        }
    }
    int src = myrank - myrank%num_local_gpus;
    NCCLCHECK(ncclRecv(recv_buff, size, type, src, comm, stream));
    GroupEnd();
}
void _ncclAllToAll(const void *sendbuff, void *recvbuff, int size,
				   int datatype, ncclComm_t comm,
				   hipStream_t stream, int num_of_peers) {
	ncclDataType_t type = _get_proper_datatype(datatype);
	int unit_size = sizeof(type);

	switch(type){
		case 0:
		case 1:
			unit_size = 1;
			break;
		case 5:
			unit_size = 2;
			break;
		case 2:
		case 3:
		case 7:
			unit_size = 4;
			break;
		case 4:
		case 8:
			unit_size = 8;
			break;
	}

	int split_size = size / num_of_peers;
	GroupStart();
	for(int i = 0; i < num_of_peers; i++){
		NCCLCHECK(ncclSend(sendbuff+i*split_size*unit_size, split_size, type, i, comm, stream));
		NCCLCHECK(ncclRecv(recvbuff+i*split_size*unit_size, split_size, type, i, comm, stream));
	}
	GroupEnd();
}

void _ncclBroadcast(const void *sendbuff, void *recvbuff, int size,
                    int datatype, int root, ncclComm_t comm,
                    hipStream_t stream) {
    NCCLCHECK(ncclBroadcast((const void *)sendbuff, (void *)recvbuff, size,
                            _get_proper_datatype(datatype), root, comm,
                            stream));
}

void _ncclAllGather(const void *sendbuff, void *recvbuff, int size,
                    int datatype, ncclComm_t comm, hipStream_t stream) {
    NCCLCHECK(ncclAllGather((const void *)sendbuff, (void *)recvbuff, size,
                            _get_proper_datatype(datatype), comm, stream));
}

void _ncclReduceScatter(const void *sendbuff, void *recvbuff, int size,
                        int datatype, int op, ncclComm_t comm,
                        hipStream_t stream) {
    NCCLCHECK(ncclReduceScatter((const void *)sendbuff, (void *)recvbuff, size,
                                _get_proper_datatype(datatype),
                                _get_proper_redop(op), comm, stream));
}

void _ncclSend(const void *sendbuff, int size, int datatype, int target,
               ncclComm_t comm, hipStream_t stream) {
    NCCLCHECK(ncclSend(sendbuff, size, _get_proper_datatype(datatype), target,
                       comm, stream));
}

void _ncclRecv(void *recvbuff, int size, int datatype, int src, ncclComm_t comm,
               hipStream_t stream) {
    NCCLCHECK(ncclRecv(recvbuff, size, _get_proper_datatype(datatype), src,
                       comm, stream));
}

void dlarrayAllReduce(DLArray *input_array, DLArray *output_array, int datatype,
                      int op, ncclComm_t comm, DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < input_array->ndim; i++) {
        size = size * input_array->shape[i];
    }
    float *input_data_buffer = (float *)(input_array->data);
    float *output_data_buffer = (float *)(output_array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclAllReduce(input_data_buffer, output_data_buffer, size, datatype, op,
                   comm, stream);
}

void dlarrayReduce(DLArray *input_array, DLArray *output_array, int datatype,
                   int op, int root, ncclComm_t comm,
                   DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < input_array->ndim; i++) {
        size = size * input_array->shape[i];
    }
    float *input_data_buffer = (float *)(input_array->data);
    float *output_data_buffer = (float *)(output_array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclReduce(input_data_buffer, output_data_buffer, size, datatype, op, root,
                comm, stream);
}

void dlarrayBroadcast(DLArray *input_array, DLArray *output_array, int datatype,
                      int root, ncclComm_t comm, DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < input_array->ndim; i++) {
        size = size * input_array->shape[i];
    }
    float *input_data_buffer = (float *)(input_array->data);
    float *output_data_buffer = (float *)(output_array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclBroadcast(input_data_buffer, output_data_buffer, size, datatype, root,
                   comm, stream);
}

void dlarrayAllGather(DLArray *array, DLArray *output_array, int datatype,
                      ncclComm_t comm, DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < array->ndim; i++) {
        size = size * array->shape[i];
    }
    int output_size = 1;
    for (int i = 0; i < output_array->ndim; i++) {
        output_size = output_size * output_array->shape[i];
    }
    float *input_buffer = (float *)(array->data);
    float *output_buffer = (float *)(output_array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclAllGather(input_buffer, output_buffer, size, datatype, comm, stream);
}

void dlarrayReduceScatter(DLArray *array, DLArray *output_array, int datatype,
                          int op, ncclComm_t comm,
                          DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < output_array->ndim; i++) {
        size = size * output_array->shape[i];
    }
    float *input_buffer = (float *)(array->data);
    float *output_buffer = (float *)(output_array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclReduceScatter(input_buffer, output_buffer, size, datatype, op, comm,
                       stream);
}

void dlarrayAllToAll(DLArray *sendarray, DLArray *recvarray, int datatype, ncclComm_t comm,
					 DLStreamHandle stream_handle, int num_of_peers){
	int size = 1;
	for(int i = 0; i < sendarray->ndim; i++){
		size = size * sendarray->shape[i];
	}
	float *send_data_buffer = (float *)(sendarray->data);
	float* recv_data_buffer = (float *)(recvarray->data);
	hipStream_t stream = *(hipStream_t *)stream_handle->handle;

	_ncclAllToAll(send_data_buffer, recv_data_buffer, size, datatype, comm, stream, num_of_peers);
}

void dlarrayHAllToAll(DLArray *sendarray, DLArray *recvarray, int datatype, ncclComm_t comm, DLStreamHandle stream_handle, int num_nodes, int num_local_gpus){
    int size = 1;
    for(int i = 0; i < sendarray->ndim; i++){         
        size = size * sendarray->shape[i];                      
    }            
    float *send_data_buffer = (float *)(sendarray->data); 
    float* recv_data_buffer = (float *)(recvarray->data);                    
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
                        
    _ncclHAllToAll(send_data_buffer, recv_data_buffer, size, datatype, comm, stream, num_nodes, num_local_gpus);
}


void dlarraySend(DLArray *array, int datatype, int target, ncclComm_t comm,
                 DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < array->ndim; i++) {
        size = size * array->shape[i];
    }
    float *data_buffer = (float *)(array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;

    _ncclSend(data_buffer, size, datatype, target, comm, stream);
}

void dlarrayRecv(DLArray *array, int datatype, int src, ncclComm_t comm,
                 DLStreamHandle stream_handle) {
    int size = 1;
    for (int i = 0; i < array->ndim; i++) {
        size = size * array->shape[i];
    }
    float *data_buffer = (float *)(array->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;

    _ncclRecv(data_buffer, size, datatype, src, comm, stream);
}

void dlarrayHA2AGather(DLArray *sendarr, DLArray *recvarr, int datatype, int myrank, int num_local_gpus, ncclComm_t comm, DLStreamHandle stream_handle){
    int size = 1;
    for (int i = 0; i < sendarr->ndim; i++){
        size *= sendarr->shape[i];
    }
    float* recv_buff = (float*)(recvarr->data);
    float* send_buff = (float*)(sendarr->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclHA2AGather(send_buff, recv_buff, size, datatype, myrank, num_local_gpus, comm, stream);
}


void dlarrayHA2AScatter(DLArray *sendarr, DLArray *recvarr, int datatype, int myrank, int num_local_gpus, ncclComm_t comm, DLStreamHandle stream_handle){
    int size = 1;
    for (int i = 0; i < recvarr->ndim; i++){
        size *= recvarr->shape[i];
    }
    float* recv_buff = (float*)(recvarr->data);
    float* send_buff = (float*)(sendarr->data);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    _ncclHA2AScatter(send_buff, recv_buff, size, datatype, myrank, num_local_gpus, comm, stream);
}

void commDestroyNccl(ncclComm_t *comm) {
    NCCLCHECK(ncclCommDestroy(*comm));
}

void display(const float *device_data, int dev_id, int size) {
    printf("Display Device %d:\n", dev_id);
    CUDACHECK(hipSetDevice(dev_id));
    float *host_buff;
    CUDACHECK(
        hipHostAlloc(&host_buff, size * sizeof(float), hipHostMallocDefault));
    CUDACHECK(hipMemcpy(host_buff, device_data, size * sizeof(float),
                         hipMemcpyDeviceToHost));
    for (int i = 0; i < size; i++) {
        printf("%f ", host_buff[i]);
    }
    printf("\n");
    CUDACHECK(hipHostFree(host_buff));
}

void print_array(float *array, int size) {
    float *output;
    output = (float *)malloc(sizeof(float) * size);
    hipMemcpy(output, array, size * sizeof(float), hipMemcpyHostToHost);
    for (int i = 0; i < size; i++) {
        printf("%f ", output[i]);
    }
    printf("\n");
}
